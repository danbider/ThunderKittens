#include "hip/hip_runtime.h"
# include "src/kittens.cuh"
#include <cuda/pipeline>

#define NUM_WORKERS (16) // hardcoded, don't change
#define NUM_THREADS (NUM_WORKERS*kittens::WARP_THREADS)
#define D_QK (256)       // hardcoded, don't change
#define D_VO (64)        // hardcoded but can be changed with some effort

using namespace kittens;

using layout = kittens::ducks::st_layout::swizzle;

// sum of an array of tiles -- in fp32 to preserve maximal accuracy
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void tile_reduce(ST &dst, const ST (&src)[N_TILES]) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;
    constexpr int RESPONSIBLE_ELEMENTS = (ST::num_elements+STRIDE-1) / STRIDE; // we know in advance this divides evenly.
    float acc[RESPONSIBLE_ELEMENTS];
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] = __bfloat162float(dst.data[idx]); // start
    }
    // then propagate accumulation through
    for(int i = 0; i < N_TILES; i++) {
        #pragma unroll
        for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
            int idx = threadIdx.x + j*STRIDE;
            if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] += __bfloat162float(src[i].data[idx]); // accumulate
        }
    }
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) dst.data[idx] = __float2bfloat16(acc[j]); // set
    }
}


// alternatively, sum onto the FIRST tile -- needed by attention.
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void tile_reduce(ST (&dst)[N_TILES]) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;
    constexpr int RESPONSIBLE_ELEMENTS = (ST::num_elements+STRIDE-1) / STRIDE; // we know in advance this divides evenly.
    float acc[RESPONSIBLE_ELEMENTS];
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] = __bfloat162float(dst[0].data[idx]); // start
    }
    // then propagate accumulation through
    for(int i = 1; i < N_TILES; i++) {
        #pragma unroll
        for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
            int idx = threadIdx.x + j*STRIDE;
            if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] += __bfloat162float(dst[i].data[idx]); // accumulate
        }
    }
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) dst[0].data[idx] = __float2bfloat16(acc[j]); // set
    }
}

__global__ __launch_bounds__(NUM_THREADS, 1)
void hedgehog_linear_attention(int n, const bf16* __q, const bf16* __k, const bf16* __v, bf16* __o, bf16* __kv_state) {

    using G = kittens::group<NUM_WORKERS>;

    auto warpid = kittens::warpid();
    auto lane   = kittens::laneid();

    const bf16 *q_g   = reinterpret_cast<const bf16*>(__q)+blockIdx.x*(n*D_QK);
    const bf16 *k_g   = reinterpret_cast<const bf16*>(__k)+blockIdx.x*(n*D_QK);
    const bf16 *v_g   = reinterpret_cast<const bf16*>(__v)+blockIdx.x*(n*D_VO);
          bf16 *o_g   = reinterpret_cast<bf16*>      (__o)+blockIdx.x*(n*D_VO);
          bf16 *kv_g  = reinterpret_cast<bf16*>(__kv_state)+blockIdx.x*(D_QK*D_VO);

    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);
    using QK_BLOCK = st_bf_1x1<layout>;
    using VO_BLOCK = st_bf_1x4<layout>;
    QK_BLOCK (&q_s)[2][NUM_WORKERS] = al.allocate<QK_BLOCK, 2, NUM_WORKERS>(); // 2 * 8192 bytes -- 16x256
    QK_BLOCK (&k_s)[2][NUM_WORKERS] = al.allocate<QK_BLOCK, 2, NUM_WORKERS>(); // 2 * 8192 bytes -- 16x256
    VO_BLOCK (&v_s)[2]              = al.allocate<VO_BLOCK, 2>(); // 2 * 2048 bytes
    VO_BLOCK (&o_s)[2]              = al.allocate<VO_BLOCK, 2>(); // 2 * 2048 bytes

    // att_accumulate is not actually a QK block, even if it happens to be the same type here.
    st_bf_1x1<layout> (&att_accumulate)[NUM_WORKERS] = al.allocate<st_bf_1x1<layout>, NUM_WORKERS>(); // 8192 bytes -- 16x(16x16) = 16x(256)
    VO_BLOCK          (&kv_accumulate) [NUM_WORKERS] = al.allocate<VO_BLOCK,          NUM_WORKERS>(); // 32768 bytes -- 16x(16x64)

    rt_fl_1x4<> kv_state; // kv state gets propagated through here, split among all 16 workers.

    zero(kv_state);       // everyone zeroes their part of the kv state.

    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> qkv_barrier;
    if (threadIdx.x == 0) {init(&qkv_barrier, NUM_THREADS);}
    __syncthreads();
    load_async(q_s[0][warpid], q_g + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
    load_async(k_s[0][warpid], k_g + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
    G::load_async(v_s[0],      v_g, D_VO, qkv_barrier); // just collaboratively load v

    int n_blocks = n / kittens::TILE_DIM; // SA: should denom just be NUM_WORKERS

    int tic = 0, toc = 1;
    for(int block = 0; block < n_blocks; block++, tic^=1, toc^=1) {
        rt_bf_1x1<> q, k, local_attn_bf; // Stores for each of 16 tokens, 16 features of the D_QK per warp
        rt_fl_1x1<> local_attn;          
        rt_bf_1x4<> v;                   // 16 x 64, 16 tokens with 64 head_dim each
        rt_fl_1x4<> o;

        // load new q, k, v into shared memory and zero o -- collaboratively, across the whole group
        // (the reason to do this is to fill larger transactions.)
        qkv_barrier.arrive_and_wait();
        if(block+1 < n_blocks) {
            load_async(q_s[toc][warpid], q_g + (block+1)*NUM_WORKERS*QK_BLOCK::num_elements + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
            load_async(k_s[toc][warpid], k_g + (block+1)*NUM_WORKERS*QK_BLOCK::num_elements + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
            G::load_async(v_s[toc],      v_g + (block+1)*VO_BLOCK::num_elements, D_VO, qkv_barrier); // just collaboratively load v
        }

        load(q, q_s[tic][warpid]);
        load(k, k_s[tic][warpid]);
        zero(local_attn);
        mma_ABt(local_attn, q, k, local_attn);
        store(att_accumulate[warpid], local_attn);
        // sum up local attention
        __syncthreads();
        tile_reduce<NUM_WORKERS>(att_accumulate); // now sum is in the first element.
        __syncthreads();
        load(v, v_s[tic]); // everyone needs v
        auto &v_col = swap_layout_inplace(v); // prepare for MMA

        // just one worker 
        if(warpid == 0) {
            load(local_attn_bf, att_accumulate[0]);
            make_causal(local_attn_bf, local_attn_bf, kittens::base_types::constants<bf16>::zero());
            zero(o);
            mma_AB(o, local_attn_bf, v_col, o); // causal bit.
            store(o_s[tic], o);
            // we have now taken care of the current attention block
        }

        // now we use the previous recurrent KV state to finish o_s[tic]
        rt_bf_1x4<> kv_bf;
        copy(kv_bf, kv_state);
        auto &kv_bf_col = swap_layout_inplace(kv_bf);
        zero(o);
        mma_AB(o, q, kv_bf_col, o);
        store(kv_accumulate[warpid], o);
        __syncthreads();
        tile_reduce<NUM_WORKERS>(o_s[tic], kv_accumulate); // sum onto o_s.

        // we've now successfully compute o_s[tic] -- we can store it.
        __syncthreads();
        G::store(o_g + block*VO_BLOCK::num_elements, o_s[tic], D_VO); // output is B H N D_VO

        // finally we need to update the kv state for future iterations
        auto &kt = transpose_inplace(k); // k is now transposed! k has been invalidated; there is only kt.
        mma_AB(kv_state, kt, v_col, kv_state);
    }

     __syncthreads();
    store(kv_g + warpid*kv_state.num_elements, kv_state, D_VO); 
    __syncthreads();
}


// #include "harness.impl"  // (comment out when using the code below)


// For binding to PyTorch (comment out include for harness.imple when using the code below)
#include "src/common/pyutils/torch_helpers.cuh"
#include <iostream>
void hedgehog_fwd_tk(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o, torch::Tensor kv_state) {
    std::cout << "Entered Hedgehog handler" << std::endl;
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(o);
    CHECK_INPUT(kv_state);
    
    auto batch = q.size(0);
    auto heads = q.size(1);
    auto threads = NUM_WORKERS * kittens::WARP_THREADS;
    auto n     = q.size(2);
    bool k_same = true, o_same = true;
    for(auto i = 0; i < 4; i++) { 
        k_same &= q.size(i) == k.size(i);
        o_same &= v.size(i) == o.size(i);
    }
    
    // This is just a restriction of what we're doing now...
    TORCH_CHECK(k_same, "Q and K should be same size");
    TORCH_CHECK(o_same, "V and O should be same size");
    TORCH_CHECK(q.scalar_type() == c10::ScalarType::BFloat16, "Q is a Bfloat");
    TORCH_CHECK(k.scalar_type() == c10::ScalarType::BFloat16, "K is a Bfloat");
    TORCH_CHECK(v.scalar_type() == c10::ScalarType::BFloat16, "V is a Bfloat");
    TORCH_CHECK(o.scalar_type() == c10::ScalarType::BFloat16, "O is a Bfloat");
    TORCH_CHECK(n % (NUM_WORKERS*kittens::TILE_DIM) == 0, "The number of elements should be divisible the number of workers times stored fragments");

    // convert to bf16
    c10::BFloat16 *q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_ptr = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_ptr = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *o_ptr = o.data_ptr<c10::BFloat16>();
    c10::BFloat16 *kv_ptr = kv_state.data_ptr<c10::BFloat16>();

    const bf16* q_bf = reinterpret_cast<const bf16*>(q_ptr);
    const bf16* k_bf = reinterpret_cast<const bf16*>(k_ptr);
    const bf16* v_bf = reinterpret_cast<const bf16*>(v_ptr);
          bf16* o_bf = reinterpret_cast<bf16*>(o_ptr);
          bf16* kv_bf = reinterpret_cast<bf16*>(kv_ptr);

    std::cout << "Checks and casts" << std::endl;
    unsigned long mem_size = kittens::MAX_SHARED_MEMORY;
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        hedgehog_linear_attention),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );

    std::cout << "Set dynamic memory" << std::endl;
    hedgehog_linear_attention<<<batch*heads,threads,mem_size>>>(n, q_bf, k_bf, v_bf, o_bf, kv_bf);

    std::cout << "Launched kernel" << std::endl;
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    std::cout << "Exiting" << std::endl;
}

