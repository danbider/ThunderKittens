#include "hip/hip_runtime.h"
// #define TORCH_COMPILE // defined by default for PyTorch bindings - to use cpp harness, comment this out

#ifdef TORCH_COMPILE
#include "src/kittens.cuh"
#else
#include "../../../src/kittens.cuh"
#endif
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#define NUM_WORKERS (8)
#define NUM_WARPGROUPS (NUM_WORKERS/(kittens::WARPGROUP_WARPS))
#define NUM_WORKERS_KV (1)

using namespace kittens;

using layout_q = kittens::ducks::st_layout::wgmma_swizzle; 
using layout_k = kittens::ducks::st_layout::wgmma_swizzle;
using layout_v = kittens::ducks::st_layout::wgmma_interleave;
using layout_o = kittens::ducks::st_layout::swizzle;

template<int D> struct fwd_attend_ker_tile_dims {
    constexpr static int tile_width = D/kittens::TILE_DIM;
    constexpr static int qo_height  = 4;
    constexpr static int kv_height  = 512/D;
};

template<int D>
__global__  __launch_bounds__((NUM_WORKERS)*kittens::WARP_THREADS, 2)
void fwd_attend_ker_dim(int N, const CUtensorMap* tma_q, const CUtensorMap* tma_k, const CUtensorMap* tma_v, CUtensorMap* tma_o) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    constexpr int tile_width = fwd_attend_ker_tile_dims<D>::tile_width; // constants
    constexpr int qo_height  = fwd_attend_ker_tile_dims<D>::qo_height;
    constexpr int kv_height  = fwd_attend_ker_tile_dims<D>::kv_height;

    st_bf<qo_height, tile_width, layout_q> (&q_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>,    NUM_WARPGROUPS>();
    st_bf<kv_height, tile_width, layout_k> (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_k>, 2, NUM_WORKERS_KV>();
    st_bf<kv_height, tile_width, layout_v> (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_v>, 2, NUM_WORKERS_KV>();
    st_bf<qo_height, tile_width, layout_o> (&o_smem)   [NUM_WARPGROUPS] = *reinterpret_cast<st_bf<qo_height, tile_width, layout_o>(*)[NUM_WARPGROUPS]>(q_smem); // reuse q memory

    int tic = 0, toc = 1;
 
    rt_fl<1, kv_height> att_block;
    rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, tile_width> o_prev;
    col_vec<rt_fl<1, kv_height>> max_vec_last, max_vec;
    col_vec<rt_fl<1, kv_height>> norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    int kv_blocks = N / (NUM_WORKERS_KV*k_smem[0][0].rows);

    // no more idx calculation - just instantiate an iterator and do:
    // - .load_async
    // - .arrive_and_wait
    // - .hasNext
    // - .store
    __shared__ uint64_t qsmem_barrier, ksmem_barrier, vsmem_barrier;
    TileIterator<st_bf<qo_height, tile_width, layout_q>, NUM_WARPGROUPS> Q(tma_q, gridDim.x, blockIdx.y, &qsmem_barrier); 
    TileIterator<st_bf<kv_height, tile_width, layout_k>, NUM_WORKERS_KV> K(tma_k, kv_blocks, blockIdx.y, &ksmem_barrier); 
    TileIterator<st_bf<kv_height, tile_width, layout_v>, NUM_WORKERS_KV> V(tma_v, kv_blocks, blockIdx.y, &vsmem_barrier); 
    TileIterator<st_bf<qo_height, tile_width, layout_o>, NUM_WARPGROUPS> O(tma_o, gridDim.x, blockIdx.y);

    Q.load_async(q_smem, blockIdx.x);
    K.load_async(k_smem[tic]); 
    V.load_async(v_smem[tic]); 

    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_prev);
    __syncthreads();

    Q.arrive_and_wait(); 

    if constexpr (D == 64) { warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f)); } 
    else { warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.08838834764f)); }

    while (K.hasNext()) {

        K.arrive_and_wait(); 
        V.arrive_and_wait(); 
        __syncthreads();

        if (K.hasNext()) {
            K++, K.load_async(k_smem[toc]); 
            V++, V.load_async(v_smem[toc]);
        }

        warpgroup::mma_fence(att_block);
        warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic][0]);
        warpgroup::mma_commit_group();

        copy(norm_vec_last, norm_vec);
        copy(max_vec_last,  max_vec);

        warpgroup::mma_async_wait();

        row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
        sub_row(att_block, att_block, max_vec);
        exp(att_block, att_block);

        sub(max_vec_last, max_vec_last, max_vec);
        exp(max_vec_last, max_vec_last);
        mul(norm_vec, norm_vec, max_vec_last);

        row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
        div_row(att_block, att_block, norm_vec);

        mul(norm_vec_last, norm_vec_last, max_vec_last);
        div(norm_vec_last, norm_vec_last, norm_vec);

        copy(att_block_mma, att_block); // convert to bf16 for mma
        mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it

        warpgroup::mma_fence(o_prev);
        warpgroup::mma_AB(o_prev, att_block_mma, v_smem[tic][0]);
        warpgroup::mma_commit_group();

        tic ^= 1; 
        toc ^= 1; 
    }

    warpgroup::store(o_smem[warpgroupid], o_prev); 
    __syncthreads();

    O.store_async(o_smem, blockIdx.x);
    tma::store_async_wait();
}

#ifdef TORCH_COMPILE
#include "src/common/pyutils/torch_helpers.cuh"
#include <iostream>

void attention_forward(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o) {

    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(o);

    auto batch   = q.size(0);
    auto heads   = q.size(1);
    auto N       = q.size(2);
    auto D       = q.size(3);

    auto threads = NUM_WORKERS * kittens::WARP_THREADS;

    TORCH_CHECK(q.scalar_type() == c10::ScalarType::BFloat16, "q must be bf16");
    TORCH_CHECK(k.scalar_type() == c10::ScalarType::BFloat16, "k must be bf16");
    TORCH_CHECK(v.scalar_type() == c10::ScalarType::BFloat16, "v must be bf16");
    TORCH_CHECK(o.scalar_type() == c10::ScalarType::BFloat16, "o must be bf16");

    // make sure sequence length is multiple of 128 for now
    TORCH_CHECK(N % (NUM_WORKERS * kittens::TILE_DIM) == 0, "Please pad sequence length to be multiple of 128");

    // make sure D = 64 or 128
    TORCH_CHECK(D == 64 || D == 128, "Currently, only D = 64 or 128 is supported");

    // convert to bf16
    c10::BFloat16 *q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_ptr = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_ptr = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *o_ptr = o.data_ptr<c10::BFloat16>();

    const bf16* q_bf = reinterpret_cast<const bf16*>(q_ptr);
    const bf16* k_bf = reinterpret_cast<const bf16*>(k_ptr);
    const bf16* v_bf = reinterpret_cast<const bf16*>(v_ptr);
    bf16* o_bf = reinterpret_cast<bf16*>(o_ptr);

    if (D == 64) {

        CUtensorMap* tma_q_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<64>::qo_height, fwd_attend_ker_tile_dims<64>::tile_width, layout_q>>(q_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<64>::qo_height * 16));
        CUtensorMap* tma_k_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<64>::kv_height, fwd_attend_ker_tile_dims<64>::tile_width, layout_k>>(k_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<64>::kv_height * 16));
        CUtensorMap* tma_v_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<64>::kv_height, fwd_attend_ker_tile_dims<64>::tile_width, layout_v>>(v_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<64>::kv_height * 16));
        CUtensorMap* tma_o_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<64>::qo_height, fwd_attend_ker_tile_dims<64>::tile_width, layout_o>>(o_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<64>::qo_height * 16));

        unsigned long mem_size = 112000;
        hipFuncSetAttribute(reinterpret_cast<const void*>(fwd_attend_ker_dim<64>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

        dim3 grid(N/(NUM_WORKERS*kittens::TILE_DIM), batch*heads, 1);

        fwd_attend_ker_dim<64><<<grid, threads, mem_size>>>(N, tma_q_d, tma_k_d, tma_v_d, tma_o_d);
    }
    else {
        CUtensorMap* tma_q_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<128>::qo_height, fwd_attend_ker_tile_dims<128>::tile_width, layout_q>>(q_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<128>::qo_height * 16));
        CUtensorMap* tma_k_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<128>::kv_height, fwd_attend_ker_tile_dims<128>::tile_width, layout_k>>(k_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<128>::kv_height * 16));
        CUtensorMap* tma_v_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<128>::kv_height, fwd_attend_ker_tile_dims<128>::tile_width, layout_v>>(v_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<128>::kv_height * 16));
        CUtensorMap* tma_o_d = tma::allocate_and_create_tensor_map<kittens::st_bf<fwd_attend_ker_tile_dims<128>::qo_height, fwd_attend_ker_tile_dims<128>::tile_width, layout_o>>(o_bf, (batch*heads*N)/(fwd_attend_ker_tile_dims<128>::qo_height * 16));

        unsigned long mem_size = 112000;
        hipFuncSetAttribute(reinterpret_cast<const void*>(fwd_attend_ker_dim<128>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

        dim3 grid(N/(NUM_WORKERS*kittens::TILE_DIM), batch*heads, 1);

        fwd_attend_ker_dim<128><<<grid, threads, mem_size>>>(N, tma_q_d, tma_k_d, tma_v_d, tma_o_d);
    }
    
    CHECK_CUDA_ERROR(hipGetLastError());
}
#else
#include "harness_h100_fwd.impl"
#endif