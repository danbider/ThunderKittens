#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <assert.h>
#include <mma.h>
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>
using namespace nvcuda;

# include "src/kittens.cuh"
// # include "src/common/pyutils/torch_helpers.cuh"

using namespace kittens;

// Types
typedef rt_bf<1, 1> _rtd_qk;
typedef rt_bf<1, 4> _rtd_v;
typedef rt_fl<1, 1> _rtd_qk_accum;
typedef rt_fl<1, 4> _rtd_v_accum;
typedef rt_bf<1, 1, ducks::rt_layout::col> _rtd_qk_col;
typedef rt_bf<1, 4, ducks::rt_layout::col> _rtd_v_col;

#define N_WARPS 4
#define WARP_SIZE 32
#define N_THREADS (N_WARPS * WARP_SIZE)

/*
int row = ???;
for(int i = 0; i < cols; i+=kittens::WARP_THREADS) tile[{row, i}] = 0;

*/

// Compute A0.
// We are computing V.cumsum(dim=0) in this example (Across the sequence)
// We first compute the local cumulative sum.
// Each has their local copy of V, we have to add in two elements
// 1. the preceding a0 from the last iteration (Stored in total_a0)
// 2. We need to compute a cumulative sum across these tiles.
// To handle 1, we add in total_a0 to a0
// TESTED AND CONFIRMED WORKING
__device__
void tb_cumsum(
    st_bf_1x4<ducks::st_layout::xor_swizzle> (&dst)[N_WARPS], 
    st_bf_1x4<ducks::st_layout::xor_swizzle>::row_vec &total, // tile should have same width as the tile and same type.
    const st_bf_1x4<ducks::st_layout::xor_swizzle> (&src)[N_WARPS]
) {
    using T = st_bf_1x4<ducks::st_layout::xor_swizzle>;
    using H = T::dtype;

    const int width = T::width;
    const int height = T::height;
    const int rows = T::rows;
    const int row_stride = T::cols;
    
    // Threads are assigned to cols, and then go sequentially through the all rows in the warps
    __syncthreads();
    for(int col = threadIdx.x; col < dst[0].cols; col+= N_THREADS) {
        // this is resonsible for this column value.
        H v = total.data[col];
        for(auto w = 0; w < N_WARPS; w++) {
            for(int r = 0; r < rows; r++) {
                v += src[w][int2{r,col}];
                dst[w][int2{r,col}] = v;
            }
        } 
        total.data[col] = v;  
    } 
}


// We write the local copy, and we want to compute a cumulative sum:
// 1. we need to add in the A0 that we computed in the last loop (handled by warp adding to its copy)
// 2. we need the A1 fragments computed from the preceding warp.
// Then a1 has the "preceding" a1 for each warp; total_a1 is the next stage of what we need to build.
template<int NUM_WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void cumsum_inplace(ST (&x)[N_TILES], ST &total) {
    constexpr int STRIDE = NUM_WORKERS*kittens::WARP_THREADS;
    for(int i = 1; i < N_TILES; i++) {
        for(int j = threadIdx.x; j < ST::num_elements; j+=STRIDE) {
            x[i].data[j] += x[i-1].data[j];
        }
    }
    __syncthreads();
    for(int j = threadIdx.x; j < ST::num_elements; j+=STRIDE) {
        total.data[j] += x[N_TILES-1].data[j];
    }
    __syncthreads();
}

__device__
void reduce_tile_tiles(
    st_bf_1x4<ducks::st_layout::xor_swizzle> &dst, 
    const st_bf_1x4<ducks::st_layout::xor_swizzle> (&src)[N_WARPS] 
) {
    using T = st_bf_1x4<ducks::st_layout::xor_swizzle>;
    using TT = T::dtype;
    auto col = threadIdx.x % (kittens::TILE_DIM*T::width);
    auto row = threadIdx.x / (kittens::TILE_DIM*T::width); 
    __syncthreads(); 
    const int _row_stride = T::cols; // SA TODO: double check
    auto idx = row*_row_stride+col;
    assert(N_THREADS % (kittens::TILE_DIM * T::width) == 0);
    auto rows_per_block = N_THREADS / (kittens::TILE_DIM*T::width);
    auto row_skip       = rows_per_block * _row_stride;
    for(auto h = 0; h < T::height; h++) {
        for(auto rows = 0; rows < rows_per_block; rows ++, idx += row_skip) {
            int _idx = h*_row_stride*kittens::TILE_DIM + 0*kittens::TILE_DIM;

            T t = src[0];   // TODO: SA confirm this rewrite
            TT *src0 = t.data + _idx;
            TT v = src0[idx];
            for(int wrp = 1; wrp < N_WARPS; wrp++) {
                T t1 = src[wrp];
                TT *src1 = t1.data + _idx;
                v += src1[idx];
            }
            TT *dst0 = dst.data + _idx;
            dst0[idx] += v;
        } 
   }
}


__device__
static void
make_causal(_rtd_qk_accum &accum) {
    using T = _rtd_qk_accum::dtype;
    using T2 = rt_base<T, _rtd_qk_accum::layout>;
    
    // Structure of rt_tiles
    // tiles = [ [0, 1], [2, 3] ]
    // accum_top_row    = src.tiles[i][0].data[0], src.tiles[i][0].data[2];
    // accum_bottom_row = src.tiles[i][0].data[1], src.tiles[i][0].data[3];
                
    const int tile_height = _rtd_qk_accum::height;
    const int tile_width  = _rtd_qk_accum::width;
    auto lane  = kittens::laneid();
    auto row      = (lane / 4);
    auto next_row = row + 8;
    auto col      = 2*(lane % 4);
    float2 _zero{0.,0.};
    __syncwarp();
     for(auto h = 0; h < tile_height; h++){
        for(auto i = 0; i < 2; i++) {
            if(row       < col +     8*i) {
                // Activates on data[0][0].x and data[1][0].x --> Corresponds to: data[0], data[1]
                T2 t = accum.tiles[h][h];
                T _t = t.data[i];
                _t.x = 0.;
            }
            if (row       < col + 1 + 8*i) {
                // Activates on data[0][0].y and data[1][0].y --> Corresponds to: data[0], data[1]
                T2 t = accum.tiles[h][h];
                T _t = t.data[i];
                _t.y = 0.;
            }
            if (next_row  < col +     8*i) {
                // Activates on data[0][1].x and data[1][1].x --> Corresponds to: data[2], data[3]
                T2 t = accum.tiles[h][h];
                T _t = t.data[i+2];
                _t.x = 0.;
            }
            if (next_row  < col + 1 + 8*i) {
                // Activates on data[0][1].y and data[1][1].y --> Corresponds to: data[2], data[3]
                T2 t = accum.tiles[h][h];
                T _t = t.data[i+2];
                _t.y = 0.;
            }

            // Old version: SA confirm above replacement
            // if(row       < col +     8*i) {accum[h][h].data[i][0].x = 0.;} // i; 0-1 / 0-1 
            // if(row       < col + 1 + 8*i) {accum[h][h].data[i][0].y = 0.;}
            // if(next_row  < col +     8*i) {accum[h][h].data[i][1].x = 0.;}
            // if(next_row  < col + 1 + 8*i) {accum[h][h].data[i][1].y = 0.;} // occurs on 
        }
        for(auto w = h+1; w < tile_width; w++) {
            for(auto i = 0; i < 2; i++) {
                accum.tiles[h][w].data[i] = _zero;
            }
        }
    }
}

// Note chris left a comment: this is a wasteful way to do this.
__device__
static void mul_row_slice(_rtd_qk &reg, const int index) {
    using T = _rtd_qk::dtype;
    using T2 = rt_base<T, _rtd_qk::layout>;

    auto lane       = kittens::laneid();
    auto row        = lane / 4;
    auto col        = lane % 4; // * 2
    
    __syncwarp();
    for(auto col_offset = 0; col_offset < 2; col_offset++) {
        T2 v = (index < 8) ? reg.tiles[col_offset][0] : reg.tiles[col_offset][1]; // SA: likely need to update indexing
        T2 vs[4];
        #pragma unroll
        for(auto j=0; j < 4; j++) {
        //     vs[j] = __shfl_sync(0xFFFFFFFF, v, index*4 + j);
        }
        auto my_v = vs[col];
        __syncwarp();
        #pragma unroll
        for(auto i=0; i < 2; i++) {
            // reg.tiles[col_offset][i] = base_ops::mul(reg.tiles[col_offset][i],my_v);
        }
    }
}

template <typename H, typename T, bool _debug_build>
__global__
void a012_compute_ker(int n, int d, int dv, const T* __q, const T* __k, 
                                 const T* __v, T* __y, T* __a0, T* __a1, T* __a1y) {

    auto warpid = kittens::warpid();
    auto lane   = kittens::laneid();
    constexpr int NUM_WORKERS = N_WARPS;

    const H *_q   = reinterpret_cast<const H*>(__q)+blockIdx.x*(n*d);
    const H *_k   = reinterpret_cast<const H*>(__k)+blockIdx.x*(n*d);
    const H *_v   = reinterpret_cast<const H*>(__v)+blockIdx.x*(n*dv);
          H *_y   = reinterpret_cast<H*>(__y)+blockIdx.x*(n*dv);
    
    // Debugging Data structures
    H *_a0  = _debug_build ? reinterpret_cast<H*>(__a0)+blockIdx.x*(n*dv) : NULL;
    H *_a1  = _debug_build ? reinterpret_cast<H*>(__a1)+blockIdx.x*(n*dv) : NULL;
    H *_a1y = _debug_build ? reinterpret_cast<H*>(__a1y)+blockIdx.x*(n*dv) : NULL;
    
    // this is the CUDA shared memory
    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    st_bf_1x4<ducks::st_layout::xor_swizzle> (&v)[NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();
    st_bf_1x4<ducks::st_layout::xor_swizzle> (&y)[NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();
    st_bf_1x4<ducks::st_layout::xor_swizzle> (&a0)[NUM_WORKERS]   = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();

    // A0, A1, A2 (a2 is stored in register throughout)
    __shared__ st_bf_1x4<ducks::st_layout::xor_swizzle>::row_vec total_a0;

    // Contstants
    const int qk_tile_elements = _rtd_qk::num_elements;
    const int  v_tile_elements = _rtd_v::num_elements; 
    auto n_tiles  = n/kittens::TILE_DIM;
    auto n_blocks = n_tiles/NUM_WORKERS;
    assert(n_tiles % NUM_WORKERS == 0);

    if(warpid == 0) {
        zero(total_a0);
    }

    for(auto cur_block = 0; cur_block < n_blocks; cur_block++) {
        auto cur_idx = (cur_block * NUM_WORKERS + warpid);
        kittens::load(v[warpid], _v + cur_idx * v_tile_elements, dv);
        __syncthreads();

        /*********************************
        *  Compute A0
        *  A0: V.cumsum(dim=0) 
        **********************************/
        tb_cumsum(a0, total_a0, v);
        __syncthreads();
        copy(y[warpid], a0[warpid]);

        __syncthreads();
        store(_y + (cur_block * NUM_WORKERS + warpid)*v_tile_elements, y[warpid], dv);

    }
}

// template <typename H, typename T, bool _debug_build>
// __global__
// void a012_compute_ker(int n, int d, int dv, const T* __q, const T* __k, 
//                                  const T* __v, T* __y, T* __a0, T* __a1, T* __a1y) {

//     auto warpid = kittens::warpid();
//     auto lane   = kittens::laneid();
//     constexpr int NUM_WORKERS = N_WARPS;

//     const H *_q   = reinterpret_cast<const H*>(__q)+blockIdx.x*(n*d);
//     const H *_k   = reinterpret_cast<const H*>(__k)+blockIdx.x*(n*d);
//     const H *_v   = reinterpret_cast<const H*>(__v)+blockIdx.x*(n*dv);
//           H *_y   = reinterpret_cast<H*>(__y)+blockIdx.x*(n*dv);
    
//     // Debugging Data structures
//     H *_a0  = _debug_build ? reinterpret_cast<H*>(__a0)+blockIdx.x*(n*dv) : NULL;
//     H *_a1  = _debug_build ? reinterpret_cast<H*>(__a1)+blockIdx.x*(n*dv) : NULL;
//     H *_a1y = _debug_build ? reinterpret_cast<H*>(__a1y)+blockIdx.x*(n*dv) : NULL;
    
//     // this is the CUDA shared memory
//     extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
//     shared_allocator al((int*)&__shm[0]);
//     st_bf_1x1<ducks::st_layout::xor_swizzle> (&q)[2][NUM_WORKERS] = al.allocate<st_bf_1x1<ducks::st_layout::xor_swizzle>, 2, NUM_WORKERS>();
//     st_bf_1x1<ducks::st_layout::xor_swizzle> (&k)[2][NUM_WORKERS] = al.allocate<st_bf_1x1<ducks::st_layout::xor_swizzle>, 2, NUM_WORKERS>();
//     st_bf_1x4<ducks::st_layout::xor_swizzle> (&v)[2][NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, 2, NUM_WORKERS>();
//     st_bf_1x4<ducks::st_layout::xor_swizzle> (&y)[NUM_WORKERS]    = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();
//     st_bf_1x4<ducks::st_layout::xor_swizzle> (&ty)[NUM_WORKERS]   = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();
//     st_bf_1x4<ducks::st_layout::xor_swizzle> (&a0)[NUM_WORKERS]   = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();
//     st_bf_1x4<ducks::st_layout::xor_swizzle> (&a1)[NUM_WORKERS]   = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, NUM_WORKERS>();

//     // A0, A1, A2 (a2 is stored in register throughout)
//     __shared__ st_bf_1x4<ducks::st_layout::xor_swizzle>::row_vec total_a0;
//     __shared__ st_bf_1x4<ducks::st_layout::xor_swizzle> total_a1;

//     // Registers per thread for fragments
//     _rtd_qk qj0, qj1, kj0, kj1;
//     _rtd_qk qfrag, qkfrag;
//     _rtd_qk_col kfrag; 
//     _rtd_v_col A2j0, A2j1, vfrag;
//     _rtd_v_accum A2j0_accum, A2j1_accum, o_accum, qA2_accum;
    
//     // Registers for a1
//     _rtd_qk qk_a1_f;
//     _rtd_qk_accum qk_a1, temp_accum; 
//     _rtd_v_accum a1_accum, a1_out;
//     _rtd_v a1_frag;
//     _rtd_v_col a1_col_frag;
     
//     // Pipeline handlers and barriers
//     int tic = 0, toc = 1;
//     auto block = cooperative_groups::this_thread_block();
//     __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> qkv_barrier;
//     if (threadIdx.x == 0) {init(&qkv_barrier, block.size());}
//     __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> store_barrier;
//     if (threadIdx.x == 0) {init(&store_barrier, block.size());}
//     // Make sure no gets to the barrier before its initialized.
//     block.sync(); 

//     // Contstants
//     const int qk_tile_elements = _rtd_qk::num_elements;
//     const int  v_tile_elements = _rtd_v::num_elements; 
//     auto n_tiles  = n/kittens::TILE_DIM;
//     auto n_blocks = n_tiles/NUM_WORKERS;
//     assert(n_tiles % NUM_WORKERS == 0);

//     // Load in initial batches of QKV along the sequence dimension
//     // kittens::load_async(q[tic][warpid], _q + warpid*qk_tile_elements, d,  qkv_barrier);
//     // kittens::load_async(k[tic][warpid], _k + warpid*qk_tile_elements, d,  qkv_barrier);
//     // kittens::load_async(v[tic][warpid], _v + warpid*v_tile_elements , dv, qkv_barrier);
                
//     // // Set the tiles and accumulators to 0.
//     zero(A2j0);
//     zero(A2j1);
//     zero(A2j0_accum);
//     zero(A2j1_accum);
//     zero(a1_accum); 
//     if(warpid == 0) {
//         zero(total_a1); 
//         zero(total_a0);
//     }

//     for(auto cur_block = 0; cur_block < n_blocks; cur_block++) {//}, tic ^= 1, toc ^= 1) {
//         // qkv_barrier.arrive_and_wait(); 
//         // if(cur_block < n_blocks - 1) { // Kick off the next block load.
//         //     auto next_idx = (cur_block + 1)*NUM_WORKERS + warpid; 
//         //     kittens::load_async(q[toc][warpid], _q + next_idx * qk_tile_elements, d, qkv_barrier);
//         //     kittens::load_async(k[toc][warpid], _k + next_idx * qk_tile_elements, d, qkv_barrier);
//         //     kittens::load_async(v[toc][warpid], _v + next_idx * v_tile_elements, dv, qkv_barrier);
//         // } 
//         auto cur_idx = (cur_block * NUM_WORKERS + warpid);
//         kittens::load(v[tic][warpid], _v + cur_idx * v_tile_elements, dv);
//         __syncthreads();

//         /*********************************
//         *  Compute A0
//         *  A0: V.cumsum(dim=0) 
//         **********************************/
//         tb_cumsum(a0, total_a0, v[tic]);
//         __syncthreads();
//         copy(y[warpid], a0[warpid]); 

//         /*********************************
//         *  Compute A1 and A2 Diagonals
//         * 1. We multiply (QK.T) 
//         * 2. Multiply by V.
//         **********************************/
//         // load(qfrag, q[tic][warpid]);
//         // load(kfrag, k[tic][warpid]);
//         // load(vfrag, v[tic][warpid]);
//         // transpose_inplace(kfrag); 
        
//         // zero(temp_accum);
//         // zero(o_accum);
//         // mma(temp_accum, qfrag, kfrag, temp_accum);
//         // make_causal(temp_accum);
//         // // Save for the A1 term in qk_a1 so we can reuse temp_accum
//         // copy(qk_a1, temp_accum); 
        
//         // // Produce output w/r/t A1
//         // // Qc@A1 + make_causal(Qc@Ktc)@Vc
//         // copy(qk_a1_f, qk_a1);
//         // mma(o_accum, qk_a1_f, vfrag, o_accum);
        
//         // // Update the hidden KV state for A1
//         // zero(a1_accum);
//         // _rtd_qk rkfrag;
//         // swap_layout(rkfrag, kfrag);
//         // mma(a1_accum, rkfrag, vfrag, a1_accum);

//         // // Need to do a cumulative sum for A1 in different Warps
//         // store(a1[warpid], a1_accum);
//         // cumsum_inplace<N_WARPS>(a1, total_a1);  
//         // __syncthreads(); // need the writes to a1 to finish.

//         // // Put whole part of A1y into o_accum
//         // load(a1_col_frag, a1[warpid]);
//         // mma(o_accum, qfrag, a1_col_frag, o_accum);
        
//         /*********************************
//         *  Reuse of A1 computation in A2.
//         * 1. causal(QK)**2@V/2, where QK is in temp_accum
//         * 2. Add in the result to the causal(QK)@V + Q@A1 from A1 and store.
//         **********************************/
//         // mul(temp_accum, temp_accum, temp_accum); 
//         // mul(temp_accum, temp_accum, 0.5f);
//         // copy(qkfrag, temp_accum);
//         // mma(o_accum, qkfrag, vfrag, o_accum);

//         // Store everything so far
//         store(y[warpid], o_accum);

//         /*********************************
//         *  Handle remaining A2 elements
//         **********************************/
//         // This is the in-shared-mem portion We keep A2 in register spread across the warps. 
//         // Each warp has a 2 fragments of q and k and 1 fragment of v in memory.
//         // The indexing is below, but these are the outer products. 
//         // At this point, y[0].. y[read_block-1] contains the "diagonal" blocks of all the outputs.
//         // * We keep A2[2*warp], A2[2*warp+1] in register.
//         // * Each computes their local portion of Q[j,:]*Q*A2 and Stores it back in ty[warpid]
//         // This is hard-coded to A2 having dimension 16.
//         // __syncthreads();
//         // for(auto blk = 0; blk < NUM_WORKERS; blk++) { 
            
//         //     // This computes the "history": Q[j]@A2[j] for j=0,dots,15.
//         //     load(qj0, q[tic][warpid]);
//         //     copy(qj1, qj0); // faster than reloading?

//         //     // We store Q_j <- Q[:,j]*Q
//         //     // mul_col_slice(qj0[0][0], 2*warpid);
//         //     // mul_col_slice(qj1[0][0], 2*warpid+1);

//         //     // Compute qj, a2j portion
//         //     zero(qA2_accum);
//         //     mma(qA2_accum, qj0, A2j0, qA2_accum); // false means clear registers
//         //     mma(qA2_accum, qj1, A2j1, qA2_accum); // false means clear registers
//         //     mul(qA2_accum,  qA2_accum, 0.5f);
//         //     store(ty[warpid], qA2_accum);
            
//         //     // reduce_tile_tiles(y[blk], ty);   # SA: WARNING -- TAKING SO LONG TO COMPILE
//         //     __syncthreads();

//         //     // Update state for next round only needed if there is more work.
//         //     load(kj0, k[tic][blk]);
//         //     transpose_inplace(kj0); 
//         //     copy(kj1, kj0); 
//         //     // mul_row_slice(kj0[0][0], 2*warpid); 
//         //     // mul_row_slice(kj1[0][0], 2*warpid+1);

//         //     // Compute the A2[j] update and put it back in the register
//         //     load(vfrag, v[tic][blk]);
//         //     mma(A2j0_accum, kj0, vfrag, A2j0_accum);

//         //     _rtd_v copy_bf_A2j0;
//         //     copy(copy_bf_A2j0, A2j0_accum);
//         //     swap_layout(A2j0, copy_bf_A2j0);            

//         //     mma(A2j1_accum, kj1, vfrag, A2j1_accum); 
//         //     _rtd_v copy_bf_A2j1;
//         //     copy(copy_bf_A2j1, A2j1_accum);
//         //     swap_layout(A2j1, copy_bf_A2j1); 
//         // }
//         __syncthreads();
//         store(_y + (cur_block * NUM_WORKERS + warpid)*v_tile_elements, y[warpid], dv);
//     }
// }

// void
// a012_compute(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o) {
//     CHECK_INPUT(q);
//     CHECK_INPUT(k);
//     CHECK_INPUT(v);
//     CHECK_INPUT(o);
    
//     auto batch = q.size(0);
//     auto head  = q.size(1);
//     auto n     = q.size(2);
//     auto d     = q.size(3);
//     auto dv    = v.size(3);
//     bool k_same = true, o_same = true;
//     for(auto i = 0; i < 4; i++) { 
//         k_same &= q.size(i) == k.size(i);
//         o_same &= v.size(i) == o.size(i);
//     }
//     // This is just a restriction of what we're doing now...
//     TORCH_CHECK(k_same, "Q and K should be same size");
//     TORCH_CHECK(o_same, "V and O should be same size");

//     TORCH_CHECK(q.scalar_type() == c10::ScalarType::BFloat16, "Q is a Bfloat");
//     TORCH_CHECK(k.scalar_type() == c10::ScalarType::BFloat16, "K is a Bfloat");
//     TORCH_CHECK(v.scalar_type() == c10::ScalarType::BFloat16, "V is a Bfloat");
//     TORCH_CHECK(o.scalar_type() == c10::ScalarType::BFloat16, "O is a Bfloat");

//     using H = __hip_bfloat16;
//     using T = c10::BFloat16;
//     constexpr bool _debug_build = false;
//     const int workers = 8;

//     // q,k,v, and o are all double buffered
//     unsigned long mem_size  =  2*2*workers*sizeof(st_bf_1x1<ducks::st_layout::xor_swizzle>); // q, k and v are double buffered.
//                   mem_size +=    2*workers*sizeof(st_bf_1x4<ducks::st_layout::xor_swizzle>);
//                   mem_size += (workers+workers)*sizeof(st_bf_1x4<ducks::st_layout::xor_swizzle>);
//                   mem_size += 2*workers*sizeof(st_bf_1x4<ducks::st_layout::xor_swizzle>); // a0 and a1y

//     TORCH_CHECK(n % (workers*kittens::TILE_DIM) == 0, "The number of elements should be divisible the number of workers times stored fragments");
//     auto threads = workers * WARP_SIZE;
//     CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(
//              a012_compute_ker<H), T, _debug_build>,
//              hipFuncAttributeMaxDynamicSharedMemorySize, mem_size));
    
//     a012_compute_ker<H,T,false><<<batch*head,threads,mem_size>>>(n, d, dv, q.data_ptr<T>(), k.data_ptr<T>(), v.data_ptr<T>(),
//           o.data_ptr<T>(), NULL, NULL, NULL);

//     CHECK_CUDA_ERROR(hipDeviceSynchronize());
// }

#include "harness.impl"