#include "hip/hip_runtime.h"
# include "src/kittens.cuh"
#include <cuda/pipeline>

using namespace kittens;

#define NUM_WORKERS (4)
#define NUM_THREADS (NUM_WORKERS*kittens::WARP_THREADS)
#define NUM_WARPGROUPS (NUM_WORKERS/kittens::WARPGROUP_WARPS)

#define ATTN_D_QK 128 // hardcoded into this kernel
#define ATTN_D_VO 64  // hardcoded into this kernel

#define WINDOW_WIDTH (64)

#define tile_q_smem   st_bf_4x4<wgmma_swizzle_l>
#define tile_k_smem   st_bf_4x4<wgmma_swizzle_l>
#define tile_qf_smem  st_bf_4x4<wgmma_swizzle_l>
#define tile_kf_smem  st_bf_4x4<wgmma_interleave_l>
#define tile_v_smem   st_bf_4x4<wgmma_interleave_l>
#define tile_o_smem   st_bf_4x4<swizzle_l>
#define tile_kv_smem  st_bf_4x4<wgmma_interleave_l>
#define tile_kv2_smem st_bf_4x4<wgmma_swizzle_l>

// struct reciprocal_op {
//     template<typename T> static __device__ inline T op(const T &x) { return T(1.f)/x; }
// };
// template<> __device__ inline bf16_2 reciprocal_op::op<bf16_2>(const bf16_2 &x) { return __float2bfloat162_rn(1.f)/x; }
// template<ducks::rt::all T>
// __device__ static inline void reciprocal(T &dst, const T &src) {
//     unary_map<reciprocal_op, T>(dst, src);
// }


template<ducks::rv::all RV, ducks::rt::row_layout RT>
__device__ static inline void diagonal(RV &dst, const RT &src) {
    static_assert(RT::height == 1 && RT::width == 1);
    static_assert(RV::inner_dim == 1);

    int row = (kittens::laneid() / 4);
    int col = (kittens::laneid() % 4) * 2;

    int srclane = row*4 + (row/2);

    using T = base_types::packing<typename RT::dtype>::unpacked_type;

    #pragma unroll
    for(int i = 0; i < 2; i++) {
        T source_val = row%2 == 0 ? src.tiles[0][0].data[i*3].x : src.tiles[0][0].data[i*3].y; // what to put out?
        T val = packed_shfl_sync(MASK_ALL, source_val, srclane);
        if (i == 0) {
            dst.data[0][0].x = val;
        } else {
            dst.data[0][0].y = val;
        }
    }    
}

__global__ __launch_bounds__(NUM_THREADS, 1)
void hedgehog_attention(int n,
                        const CUtensorMap* tma_q,  const CUtensorMap* tma_k,
                        const CUtensorMap* tma_qf, const CUtensorMap* tma_kf,
                        const CUtensorMap* tma_v,
                        CUtensorMap* tma_o,
                        CUtensorMap* tma_kv) {

    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    tile_q_smem  (&q_smem)[2][2]  = al.allocate<tile_q_smem, 2, 2>(); // 16k * 2 (tictoc)
    tile_k_smem  (&k_smem)[2][2]  = al.allocate<tile_k_smem, 2, 2>(); // 16k * 2 (tictoc)
    tile_qf_smem (&qf_smem)[2][4] = al.allocate<tile_qf_smem, 2, 4>(); // 32k * 2 (tictoc)
    tile_kf_smem (&kf_smem)[2][4] = al.allocate<tile_kf_smem, 2, 4>(); // 32k * 2 (tictoc)
    tile_v_smem  (&v_smem)[2]     = al.allocate<tile_v_smem, 2>(); // 8k * 2 (tictoc)

    tile_kv_smem  (&kv_smem) = al.allocate<tile_kv_smem>(); // 8k
    tile_o_smem   (&o_smem)  = reinterpret_cast<tile_o_smem&>(kv_smem);
    tile_kv2_smem (&kv_smem_store)[4] = *reinterpret_cast<tile_kv2_smem(*)[4]>(q_smem[0]);

    // if(blockIdx.x == 0 && threadIdx.x == 0) printf("%llu\n", uint64_t(&kv_smem.data[0])-uint64_t(&q_smem[0][0].data[0]) + );

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    int tic = 0, toc = 1;

    // launch tma loads
    __shared__ uint64_t qkv_barrier;

    int blocks = n / (64);

    // launch tma loads 
    if (warpid == 0) {
        tma::init_barrier(qkv_barrier, 1); // barrier for q and k
        tma::set_bytes(qkv_barrier, 6*size_bytes<tile_q_smem> + 6*size_bytes<tile_k_smem> + size_bytes<tile_v_smem>);

        int tile_idx = (blockIdx.x * blocks);
        #pragma unroll
        for(int i = 0; i < 2; i++) {
            tma::load_async(q_smem[tic][i], tma_q, qkv_barrier, tile_idx, i);
            tma::load_async(k_smem[tic][i], tma_k, qkv_barrier, tile_idx, i);
        }
        #pragma unroll
        for(int i = 0; i < 4; i++) {
            tma::load_async(qf_smem[tic][i], tma_qf, qkv_barrier, tile_idx, i);
            tma::load_async(kf_smem[tic][i], tma_kf, qkv_barrier, tile_idx, i);
        }
        tma::load_async(v_smem[tic], tma_v, qkv_barrier, tile_idx);
    }

    rt_fl_1x4<> local_kv[4]; // 128 registers
    #pragma unroll
    for(int j = 0; j < 4; j++) {
        zero(local_kv[j]);
    }

    rt_fl_1x1<>::col_vec qk_diag; // for summing along the diagonal, for linear attention

    float last_max = -99999;
    rt_fl_1x1<>::col_vec max_vec;
    neg_infty(max_vec); // init low

    float last_norm = 0;
    rt_fl_1x1<>::col_vec norm_vec;
    zero(norm_vec);

    for (int block = 0; block < blocks; block++, tic^=1, toc^=1) {
        rt_fl_1x4 local_o; // 32 registers
        rt_fl_1x4 local_attn; // 32 registers
        rt_bf_1x4 local_attn_bf; // 16 registers

        tma::arrive_and_wait(qkv_barrier, tic);
        __syncthreads(); 

        if (warpid == 0) {
            tma::set_bytes(qkv_barrier, 6*size_bytes<tile_q_smem> + 6*size_bytes<tile_k_smem> + size_bytes<tile_v_smem>);
            if (block + 1 < blocks) {
                int tile_idx = (blockIdx.x * blocks) + (block + 1);
                #pragma unroll
                for(int i = 0; i < 2; i++) {
                    tma::load_async(q_smem[toc][i], tma_q, qkv_barrier, tile_idx, i);
                    tma::load_async(k_smem[toc][i], tma_k, qkv_barrier, tile_idx, i);
                }
                #pragma unroll
                for(int i = 0; i < 4; i++) {
                    tma::load_async(qf_smem[toc][i], tma_qf, qkv_barrier, tile_idx, i);
                    tma::load_async(kf_smem[toc][i], tma_kf, qkv_barrier, tile_idx, i);
                }
                tma::load_async(v_smem[toc], tma_v, qkv_barrier, tile_idx);
            }
        }

        __syncthreads();
        zero(local_attn);
        warpgroup::mma_fence(local_attn);
        #pragma unroll
        for(int j = 0; j < 2; j++) {
            warpgroup::mma_ABt(local_attn, q_smem[tic][j], k_smem[tic][j]);
            warpgroup::mma_commit_group(); 
        }
        warpgroup::mma_async_wait();

        __syncthreads();
        // now make causal
        #pragma unroll
        for(int j = 0; j < 4; j++) {
            auto &attn_subtile = reinterpret_cast<rt_fl_1x1<>&>(local_attn.tiles[0][j]);
            if (j>warpid) zero(attn_subtile);
            else if (j==warpid) make_causal(attn_subtile, attn_subtile, kittens::base_types::constants<float>::neg_infty());
        }

        row_max(max_vec, local_attn, max_vec); // get row max
        // ensure also max'd with previous linear attn, across all rows
        max(max_vec, max_vec, last_max); // now done
        sub_row(local_attn, local_attn, max_vec); // subtract max off

        exp(local_attn, local_attn); // exp attn matrix for softmax

        // take exp of max_vec -- this will become normalization for linear term
        rt_fl_1x1<>::col_vec rescale_vec;
        sub(rescale_vec, max_vec, last_max); // we need to figure out how to adjust previous norm
        exp(rescale_vec, rescale_vec); // exp of max_vec
        copy(norm_vec, last_norm); // set all with linear attention norm
        div(norm_vec, norm_vec, rescale_vec); // normalize norm_vec individually according to the new max vec

        row_sum(norm_vec, local_attn, norm_vec); // we have now accumulated the norm with max accounted for

        copy(local_attn_bf, local_attn); // now stored in bf16

        // now we're going to reset the local_attn and use it to do the linear version, which we need to compute the normalizations constants.
        __syncthreads();
        zero(local_attn);
        warpgroup::mma_fence(local_attn);
        #pragma unroll
        for(int j = 0; j < 4; j++) {
            warpgroup::mma_ABt(local_attn, qf_smem[tic][j], kf_smem[tic][j]);
            warpgroup::mma_commit_group(); 
        }
        warpgroup::mma_async_wait();
        // grab the diagonal from this, which we'll use to track the trace.
        diagonal<rt_fl_1x1<>::col_vec, rt_fl_1x1<>>(qk_diag, reinterpret_cast<rt_fl_1x1<>&>(local_attn.tiles[0][warpid])); // attn diagonal -- we will sum this onto the last_norm

        // basically, this is the linear attention component, tracking it for the next stage.
        float new_max;
        max(new_max, max_vec, last_max); // global softmax max from previous stages
        float rescale = __expf(last_max - new_max);
        last_norm *= rescale; // adjust previous trace using new max
        div(qk_diag, qk_diag, __expf(new_max)); // normalize diagonal to exp scaling, too
        sum(last_norm, qk_diag, last_norm); // add qk_diag to last_norm
        last_max = new_max;

        // if(block < 8) {
        //     if(blockIdx.x == 0 && threadIdx.x < 32) {
        //         printf("max_vec.x: %f, max_vec.y: %f\n", max_vec.data[0][0].x, max_vec.data[0][0].y);
        //     }
        //     if(blockIdx.x == 0 && threadIdx.x == 0) {
        //         printf("last_norm: %f, last_max: %f\n", last_norm, last_max);
        //     }
        // }

        zero(local_o);
        __syncthreads();

        for(int j = 0; j < 4; j++) {

            warpgroup::store(kv_smem, local_kv[j]);
            __syncthreads();

            warpgroup::mma_fence(local_o);
            warpgroup::mma_AB(local_o, qf_smem[tic][j], kv_smem);
            warpgroup::mma_commit_group();
            warpgroup::mma_async_wait();
            __syncthreads();

            warpgroup::mma_fence(local_kv[j]);
            warpgroup::mma_AtB(local_kv[j], kf_smem[tic][j], v_smem[tic]); // really AtB since k is transposed in advance
            warpgroup::mma_commit_group();
            warpgroup::mma_async_wait();
            __syncthreads();
        }

        div_row(local_o, local_o, norm_vec); // normalize o according to exp max for numerical stability

        __syncthreads();
        warpgroup::mma_fence(local_o);
        warpgroup::mma_AB(local_o, local_attn_bf, v_smem[tic]);
        warpgroup::mma_commit_group();
        warpgroup::mma_async_wait();

        __syncthreads(); 
        warpgroup::zero(kv_smem);
        __syncthreads(); 
        warpgroup::store(o_smem, local_o);
        __syncthreads(); 

        // launch tma store for o
        if (warpid == 0) {
            int tile_idx = (blockIdx.x * blocks) + block;
            tma::store_async(tma_o, o_smem, tile_idx);
            tma::store_commit_group(); 
        }
        __syncthreads();
        tma::store_async_wait();
        __syncthreads();
        warpgroup::zero(kv_smem);
        __syncthreads(); 
    }
    #pragma unroll
    for(int j = 0; j < 4; j++) {
        warpgroup::store(kv_smem_store[j], local_kv[j]);
        __syncthreads(); 
        if(warpid == 0) {
            tma::store_async(tma_kv, kv_smem_store[j], blockIdx.x*4 + j);
            tma::store_commit_group();
        }
        __syncthreads();
    }
    tma::store_async_wait();
}


#include "harness.impl"  // (comment out when using the code below)